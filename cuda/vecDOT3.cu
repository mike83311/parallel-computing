#include "hip/hip_runtime.h"
#include "book.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#define imin(a,b) (a<b?a:b)

const int N = 512;
const int threadsPerBlock = 512;
const int blocksPerGrid = 1;

__global__ void dot( int *a, int *b, int *c ) {
    int tid = threadIdx.x;

    if (tid < N) {
        c[tid] = a[tid] * b[tid];
    }

    __syncthreads();

    int i = N / 2;
    while (i != 0) {
        if (tid < i){
            c[tid] += c[tid + i];
        }
        __syncthreads();

        i /= 2;
    }
}


int main( void ) {
    int   *a, *b, *c;
    int   *dev_a, *dev_b, *dev_c;
    struct timespec t_start, t_end;
    int i;
    // allocate memory on the cpu side
    a = (int*)malloc( N*sizeof(int) );
    b = (int*)malloc( N*sizeof(int) );
    c = (int*)malloc( sizeof(int) );

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a,
                              N*sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b,
                              N*sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c,
                              N*sizeof(int) ) );

    // fill in the host memory with data
    srand(time(NULL));
    for (i=0; i<N; i++) {
        a[i] = rand()%256;
        b[i] = rand()%256;
    }
    // Get start time event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N*sizeof(int),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N*sizeof(int),
                              hipMemcpyHostToDevice ) ); 

    
    dot<<<blocksPerGrid,threadsPerBlock>>>( dev_a, dev_b, dev_c );
    

    
    //check cuda error
    hipError_t status = hipGetLastError();
    if ( hipSuccess != status ){
        fprintf(stderr, "Error: %s\n", hipGetErrorString(status));
        exit(1) ;
    }                                           

    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( c, dev_c,
                              sizeof(int),
                              hipMemcpyDeviceToHost ) );
    // Get stop time event    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 
    // Compute execution time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
     
    //printf("GPU result is %d\n",c);
    // start time
    clock_gettime( CLOCK_REALTIME, &t_start);
    /*CPU version*/
    int dot=0;
    for(i=0;i<N;i++){
       dot+=a[i]*b[i];
    }
    // stop time
    clock_gettime( CLOCK_REALTIME, &t_end);

    // compute and print the elapsed time in millisec
    elapsedTime = (t_end.tv_sec - t_start.tv_sec) * 1000.0;
    elapsedTime += (t_end.tv_nsec - t_start.tv_nsec) / 1000000.0;
    printf("CPU time: %13lf ms\n", elapsedTime);
    printf("CPU result is %d\n",dot);
    printf("GPU result is %d\n",*c);    
    if(*c == dot)
       printf("test pass!\n");
    else
       printf("test fail!\n");
    
    // free memory on the gpu side
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );

    // free memory on the cpu side
    free( a );
    free( b );
    free( c );
}
