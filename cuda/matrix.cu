#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define WIDTH 32
#define ROW_M 256
#define COL_M 256
#define ROW_N 256
#define COL_N 256
#define ROW_MxN ROW_M
#define COL_MxN COL_N


__global__ void MatMulKernel(float *Md, float *Nd, float *Pd, int width);
void MatMul(float *M, float *N, float *P, int width);
void printTwoDimDynamicArray(float *Array, const int col, const int row);


int main(int argc, char *argv[])
{
    int i, j, k;
    int width = WIDTH;

    // Dynamic
    float *M = (float *)malloc(ROW_M * COL_M * sizeof(float));
    float *N = (float *)malloc(ROW_N * COL_N * sizeof(float));  
    float *P = (float *)malloc(ROW_MxN * COL_MxN * sizeof(float));
    float *MxN = (float *)malloc(ROW_MxN * COL_MxN * sizeof(float));

    int pass = 1;
    
    // Initial
    for (i = 0; i < ROW_M; ++i) {
        for (j = 0; j < COL_M; ++j) {
            M[i*COL_M + j] = rand() % 5;
        }
    }
    for (i = 0; i < ROW_N; ++i) {
        for (j = 0; j < COL_N; ++j) {
            N[i*COL_N + j] = rand() % 5;
        }
    }


    struct timeval starttime, endtime;
    gettimeofday(&starttime, NULL);

    // CPU
    for (i = 0; i < ROW_M; ++i) {
        for (j = 0; j < COL_N; ++j) {
            for (k = 0; k < COL_M; ++k) {
                MxN[i*COL_N + j] += M[i*COL_M + k] * N[k*COL_N + j];
            }
        }
    }

    gettimeofday(&endtime, NULL);
    double executime;
    executime = (endtime.tv_sec - starttime.tv_sec) * 1000.0;
    executime += (endtime.tv_usec - starttime.tv_usec) / 1000.0;
    printf("CPU time: %13lf msec\n", executime);
    /*
    printTwoDimDynamicArray(M, COL_M, ROW_M);
    printf("============================\n");
    printTwoDimDynamicArray(N, COL_N, ROW_N);
    printf("============================\n");
    printTwoDimDynamicArray(MxN, COL_MxN, ROW_MxN);
    */

    // GPU
    MatMul((float *)M, (float *)N, (float *)P, width);
    
    // Compare
    for(i = 0; i < ROW_MxN; i++) {
        for(j = 0; j < COL_MxN; j++) {
            if(MxN[i*COL_MxN + j] != P[i*COL_MxN + j]) {
                printf("MxN[%d][%d] = %2.0f   P[%d][%d] = %2.0f\n", i, j, MxN[i*COL_MxN + j], i, j, P[i*COL_MxN + j]);
                pass = 0;
            }
            
        }
    }

    free(M);
    free(N);
    free(P);
    free(MxN);
    
    
    printf("Test %s\n", (pass)?"PASSED":"FAILED");
    
    return 0;
}

void printTwoDimDynamicArray(float *Array, const int col, const int row) {
    int x, y;
    for(y = 0; y != row; ++y) {
        for(x = 0; x != col; ++x)
            printf("%f ", Array[y*col + x]);
            
        printf("\n");      
    }
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(float *Md, float *Nd, float *Pd, int width){
    int row = (blockIdx.y * blockDim.y) + threadIdx.y;
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    float Pvalue = 0;
    
    // Multiply M and N
    if(row < ROW_MxN && col < COL_MxN){
        for (int k = 0; k < COL_M; ++k) {
            float Melement = *(Md + row*COL_M + k);
            float Nelement = *(Nd + k*COL_N + col);
            Pvalue += Melement * Nelement;
        }

        *(Pd + row*COL_N + col) = Pvalue;
    }
}

// Matrix multiplication - Host code
void MatMul(float *M, float *N, float *P, int width)
{
    size_t size_M = ROW_M * COL_M * sizeof(float);
    size_t size_N = ROW_N * COL_N * sizeof(float);
    size_t size_P = ROW_MxN * COL_MxN * sizeof(float);

    float *Md, *Nd, *Pd;
    
    // Allocate and Load M, N to device memory
    hipMalloc((void **)&Md, size_M);
    hipMemcpy(Md, M, size_M, hipMemcpyHostToDevice);
    
    hipMalloc((void **)&Nd, size_N);
    hipMemcpy(Nd, N, size_N, hipMemcpyHostToDevice);
    
    // Allocate P on the device
    hipMalloc((void **)&Pd, size_P);
    
    // Setup the execution configuration
    int gridDim_X = (ROW_MxN + 31)/32;
    int gridDim_Y = (COL_MxN + 31)/32;
    dim3 dimGrid(gridDim_X, gridDim_Y);
    dim3 dimBlock(width, width);
    printf("============================\n");

    // Get start time event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    // Invoke kernel
    MatMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, width);
    hipError_t cuda_err = hipGetLastError();
    if ( hipSuccess != cuda_err ){
        printf("before kernel call: error = %s\n", hipGetErrorString (cuda_err));
        exit(1) ;
    }
    
    // Get stop time event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // Compute execution time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    // Read P from device memory
    hipMemcpy(P, Pd, size_P, hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
}
