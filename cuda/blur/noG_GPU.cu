#include "hip/hip_runtime.h"
#include "opencv/cv.h"
#include "opencv/highgui.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define PI 3.14159265358979323846
#define WIDTH 32


// GPU
__global__ void Blur_Kernel(unsigned char *d_Blue, unsigned char *d_Green, unsigned char *d_Red, unsigned char *d_Blue_Blur, unsigned char *d_Green_Blur, unsigned char *d_Red_Blur, float *d_weightArr, int blurRadius, int length, int COL, int ROW);
void gaussian_blur(unsigned char *Blue, unsigned char *Green, unsigned char *Red, unsigned char *Blue_Blur, unsigned char *Green_Blur, unsigned char *Red_Blur, float *weightArr, int blurRadius, int length, int COL, int ROW);


// Weight Matrix
void createWeightMatrix(int blurRadius);
float getWeight(int blurRadius, float sigma, int x, int y);
void calculateWeightMatrix(float *weightArr, int blurRadius, float sigma);
void getFinalWeightMatrix(float *weightArr, int blurRadius);
void printArray(float *Array, int ROW, int COL);
void printArrayChar(unsigned char *Array, int ROW, int COL);



int main(int argc, char **argv){
    IplImage* sourceImg;
    IplImage* outputImg;
    

    if((sourceImg = cvLoadImage(argv[1], 1)) == NULL){
        printf("%s cannot be openned\n", argv[1]);
        exit(1);
    }
    printf("height of sourceImg: %d\n", sourceImg->height);
    printf("width of sourceImg: %d\n", sourceImg->width);
    printf("size of sourceImg: %d\n", sourceImg->imageSize);

    outputImg = cvLoadImage(argv[1], 1);



    int blurRadius = atoi(argv[2]);
    float sigma = atof(argv[3]);
    int length = blurRadius * 2 + 1;
    float *weightArr = (float *)malloc((blurRadius * 2 + 1) * (blurRadius * 2 + 1) * sizeof(float));

    calculateWeightMatrix(weightArr, blurRadius, sigma);
    getFinalWeightMatrix(weightArr, blurRadius);
    

    int COL_Step = sourceImg->widthStep;
    int COL = sourceImg->width;
    int ROW = sourceImg->height;

    // Input BGR Array
    unsigned char *Blue         = (unsigned char *)malloc(ROW * COL * sizeof(unsigned char));
    unsigned char *Green        = (unsigned char *)malloc(ROW * COL * sizeof(unsigned char));
    unsigned char *Red          = (unsigned char *)malloc(ROW * COL * sizeof(unsigned char));
    
    // Output BGR Array
    unsigned char *Blue_Blur    = (unsigned char *)malloc(ROW * COL * sizeof(unsigned char));
    unsigned char *Green_Blur   = (unsigned char *)malloc(ROW * COL * sizeof(unsigned char));
    unsigned char *Red_Blur     = (unsigned char *)malloc(ROW * COL * sizeof(unsigned char));


    // 
    for(int i = 0; i < ROW; i++){
        for(int j = 0; j < COL_Step; j = j + 3){
            Blue[i * COL + (j/3)]   = sourceImg->imageData[i * COL_Step + j];
            Green[i * COL + (j/3)]  = sourceImg->imageData[i * COL_Step + j + 1];
            Red[i * COL + (j/3)]    = sourceImg->imageData[i * COL_Step + j + 2];
        }
    }



    // GPU function
    gaussian_blur(Blue, Green, Red, Blue_Blur, Green_Blur, Red_Blur, weightArr, blurRadius, length, COL, ROW);
    
    
    // Set BGR To Output Image
    for(int i = 0; i < ROW; i++){
        for(int j = 0; j < COL_Step; j = j + 3){
            outputImg->imageData[i * COL_Step + j]     = Blue_Blur[i * COL + (j/3)];
            outputImg->imageData[i * COL_Step + j + 1] = Green_Blur[i * COL + (j/3)];
            outputImg->imageData[i * COL_Step + j + 2] = Red_Blur[i * COL + (j/3)];
        }
    }
    
    cvSaveImage("noG_black_cat.jpg", outputImg, 0);
    // cvSaveImage("output2.jpg", outputImg, 0);
    cvShowImage("sourceImg", sourceImg);
    cvShowImage("GPU", outputImg);
    
    cvWaitKey(0);
    
    cvDestroyWindow("sourceImg");
    cvReleaseImage(&sourceImg);
    cvDestroyWindow("GPU");
    cvReleaseImage(&outputImg);
    
    free(Blue);
    free(Green);
    free(Red);

    free(Blue_Blur);
    free(Green_Blur);
    free(Red_Blur);
    
    free(weightArr);


    return 0;
}





__global__ void Blur_Kernel(unsigned char *d_Blue, unsigned char *d_Green, unsigned char *d_Red, unsigned char *d_Blue_Blur, unsigned char *d_Green_Blur, unsigned char *d_Red_Blur, float *d_weightArr, int blurRadius, int length, int COL, int ROW){
    int row = (blockIdx.y * blockDim.y) + threadIdx.y;
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int i, j;


    float BBB = 0;
    float GGG = 0;
    float RRR = 0;


    if(row < ROW && col < COL){
        for(j = 0; j < length; j++){
            for(i = 0; i < length; i++){
                if(((row - blurRadius + j) < 0) || ((col - blurRadius + i) < 0) || ((row - blurRadius + j) >= ROW) || ((col - blurRadius + i) >= COL)){
                    // do nothing
                }
                else {
                    BBB += ((float)d_Blue[(row - blurRadius + j) * COL + (col - blurRadius + i)] * d_weightArr[j * length + i]);
                    GGG += ((float)d_Green[(row - blurRadius + j) * COL + (col - blurRadius + i)] * d_weightArr[j * length + i]);
                    RRR += ((float)d_Red[(row - blurRadius + j) * COL + (col - blurRadius + i)] * d_weightArr[j * length + i]);
                }

                // if(((col - blurRadius + j) >= 0) && ((row - blurRadius + i) >= 0) && ((col - blurRadius + j) < COL) && ((row - blurRadius + i) < ROW)){
                //     BBB += ((float)d_Blue[(row - blurRadius + j) * COL + (col - blurRadius + i)] * d_weightArr[j * length + i]);
                //     GGG += ((float)d_Green[(row - blurRadius + j) * COL + (col - blurRadius + i)] * d_weightArr[j * length + i]);
                //     RRR += ((float)d_Red[(row - blurRadius + j) * COL + (col - blurRadius + i)] * d_weightArr[j * length + i]);
                // }
            }
        }
        d_Blue_Blur[row * COL + col] = (unsigned char)BBB;
        d_Green_Blur[row * COL + col] = (unsigned char)GGG;
        d_Red_Blur[row * COL + col] = (unsigned char)RRR;  
    }
}

void gaussian_blur(unsigned char *Blue, unsigned char *Green, unsigned char *Red, unsigned char *Blue_Blur, unsigned char *Green_Blur, unsigned char *Red_Blur, float *weightArr, int blurRadius, int length, int COL, int ROW){
    size_t size_BGR = ROW * COL * sizeof(unsigned char);
    size_t size_weight = length * length * sizeof(float);

    unsigned char *d_Blue, *d_Green, *d_Red, *d_Blue_Blur, *d_Green_Blur, *d_Red_Blur;
    float *d_weightArr;

    // Allocate
    hipMalloc((void **)&d_Blue, size_BGR);
    hipMemcpy(d_Blue, Blue, size_BGR, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_Green, size_BGR);
    hipMemcpy(d_Green, Green, size_BGR, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_Red, size_BGR);
    hipMemcpy(d_Red, Red, size_BGR, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_weightArr, size_weight);
    hipMemcpy(d_weightArr, weightArr, size_weight, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_Blue_Blur, size_BGR);
    hipMalloc((void **)&d_Green_Blur, size_BGR);
    hipMalloc((void **)&d_Red_Blur, size_BGR);


    // Setup
    int gridDim_X = (COL + 31)/32;
    int gridDim_Y = (ROW + 31)/32;
    dim3 dimGrid(gridDim_X, gridDim_Y);
    dim3 dimBlock(WIDTH, WIDTH);

    // Get start time event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Invoke kernel
    Blur_Kernel<<<dimGrid, dimBlock>>>(d_Blue, d_Green, d_Red, d_Blue_Blur, d_Green_Blur, d_Red_Blur, d_weightArr, blurRadius, length, COL, ROW);


    hipError_t cuda_err = hipGetLastError();
    if ( hipSuccess != cuda_err ){
        printf("before kernel call: error = %s\n", hipGetErrorString (cuda_err));
        exit(1) ;
    }


    // Get stop time event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);


    // Compute execution time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    

    // Read P from device memory
    hipMemcpy(Blue_Blur, d_Blue_Blur, size_BGR, hipMemcpyDeviceToHost);
    hipMemcpy(Green_Blur, d_Green_Blur, size_BGR, hipMemcpyDeviceToHost);
    hipMemcpy(Red_Blur, d_Red_Blur, size_BGR, hipMemcpyDeviceToHost);

    
    // Free device memory
    hipFree(d_Blue);
    hipFree(d_Green);
    hipFree(d_Red);

    hipFree(d_Blue_Blur);
    hipFree(d_Green_Blur);
    hipFree(d_Red_Blur);

    hipFree(d_weightArr);

}





float getWeight(int blurRadius, float sigma, int x, int y){
    //float sigma = 5.0;//(blurRadius * 2 + 1) / 2;
    float weight = (1 / (2 * PI * sigma * sigma)) * exp(-(x * x + y * y)/(2 * sigma * sigma));
    return weight;
}

void calculateWeightMatrix(float *weightArr, int blurRadius, float sigma){
    int length = blurRadius * 2 + 1;
    for(int i = 0; i < length; i++){ 
        for(int j = 0; j < length; j++){ 
            weightArr[i * length + j] = 0.0;//getWeight(blurRadius, sigma, j - blurRadius, blurRadius - i); 
        } 
    }
}

void getFinalWeightMatrix(float *weightArr, int blurRadius){
    int length = blurRadius * 2 + 1;
    float weightSum = 0; 
    for(int i = 0; i < length; i++){ 
        for(int j = 0; j < length; j++){ 
            weightSum += weightArr[i * length + j]; 
        } 
    } 
    for(int i = 0; i < length; i++){ 
        for(int j = 0; j < length; j++ ){ 
            weightArr[i * length + j] = (float) 1 / (length * length);//weightArr[i * length + j] / weightSum; 
        } 
    }
}

void printArrayChar(unsigned char *Array, int ROW, int COL){
    int x, y;
    for(y = 0; y != ROW; ++y) {
        for(x = 0; x != COL; ++x)
            printf("%u ", Array[y * COL + x]);
            
        printf("\n");      
    }
    printf("==============================================================\n");
}

void printArray(float *Array, int ROW, int COL){
    int x, y;
    for(y = 0; y != ROW; ++y) {
        for(x = 0; x != COL; ++x)
            printf("%lf ", Array[y * COL + x]);
            
        printf("\n");      
    }
    printf("==============================================================\n");
}
















