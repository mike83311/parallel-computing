#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#define N 10000
 __global__ void add(int *d_a, int *d_b, int *d_c){
    d_c[blockIdx.x] = d_a[blockIdx.x] + d_b[blockIdx.x];
 }

 int main(){
    int *a, *b, *c, *gold_c;
    int *d_a, *d_b, *d_c;
    int i;
    int pass = 1;


    a = (int*)malloc(N*sizeof(int));
    b = (int*)malloc(N*sizeof(int));
    c = (int*)malloc(N*sizeof(int));    
    gold_c = (int*)malloc(N*sizeof(int));
    for(i=0; i<N; i++){
        a[i] = rand()%100;
        b[i] = rand()%100;
    }
    
    struct timespec t_start, t_end;
    double elapsedTimeCPU;
    // start time
    clock_gettime( CLOCK_REALTIME, &t_start);
    
    for(i=0; i<N; i++){
        gold_c[i] = a[i] + b[i];
    }
    // stop time
    clock_gettime( CLOCK_REALTIME, &t_end);
    // compute and print the elapsed time in millisec
    elapsedTimeCPU = (t_end.tv_sec - t_start.tv_sec) * 1000.0;
    elapsedTimeCPU += (t_end.tv_nsec - t_start.tv_nsec) / 1000000.0;
    printf("CPU elapsedTime: %lf ms\n", elapsedTimeCPU);
    
    
    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));
    
    
    hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N*sizeof(int), hipMemcpyHostToDevice);
    
    // Get start time event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    add<<<N, 1>>>(d_a, d_b, d_c);
    
    // Get stop time event    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 
    // Compute execution time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    hipMemcpy(c, d_c, N*sizeof(int), hipMemcpyDeviceToHost);
    
    for(i=0; i<N; i++){
        if(gold_c[i]!=c[i]){
            pass = 0;
            break;
        }
    }
    
    if(pass==1)
        printf("test pass!\n");
    else
        printf("error...\n");
    
    return 0;
 }
