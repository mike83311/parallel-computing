#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

const int Array = 1024 * 1024 * 8;
const int threadsPerBlock = 512;

__global__ void dot(int *d_a, int *d_b, int *d_c){
    int tid = threadIdx.x;
    int tidTemp = tid ;

    while(tidTemp < Array){
        d_c[tidTemp] = d_a[tidTemp] * d_b[tidTemp];
        tidTemp += blockDim.x;
    }

    __syncthreads();

    int i = Array / 2;
    while(i != 0){
        tid = threadIdx.x;
        while(tid < i){
            d_c[tid] += d_c[tid + i];
            tid += blockDim.x;
        }
        __syncthreads();

        i /= 2;
    }
}

int main(){
    int *a, *b, *c, *gold_c;
    int *d_a, *d_b, *d_c;
    int i;
    int pass = 1;

    a = (int*)malloc(Array*sizeof(int));
    b = (int*)malloc(Array*sizeof(int));
    c = (int*)malloc(Array*sizeof(int));
    gold_c = (int*)malloc(Array*sizeof(int));
    
    for(i = 0; i < Array; i++){
        a[i] = rand()%100;
        b[i] = rand()%100;
    }

    struct timespec t_start, t_end;
    double elapsedTimeCPU;

    clock_gettime(CLOCK_REALTIME, &t_start);

    //dot
    int sum = 0;
    for(i = 0; i < Array; i++){
        //gold_c[i] = a[i] * b[i];
        sum += a[i] * b[i];
    }

    clock_gettime(CLOCK_REALTIME, &t_end);

    elapsedTimeCPU = (t_end.tv_sec - t_start.tv_sec) * 1000.0;
    elapsedTimeCPU += (t_end.tv_nsec - t_start.tv_nsec) / 1000000.0;
    printf("CPU elapsedTime: %lf ms\n", elapsedTimeCPU);
    

    hipMalloc((void**)&d_a, Array * sizeof(int));
    hipMalloc((void**)&d_b, Array * sizeof(int));
    hipMalloc((void**)&d_c, Array * sizeof(int));
    
    hipMemcpy(d_a, a, Array*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, Array*sizeof(int), hipMemcpyHostToDevice);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    dot<<<1, threadsPerBlock>>>(d_a, d_b, d_c);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float cudaelapsedTime;
    hipEventElapsedTime(&cudaelapsedTime, start, stop);
    printf("GPU elapsedTime: %lf ms\n", cudaelapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(c, d_c, Array*sizeof(int), hipMemcpyDeviceToHost);
    
    /*for(i = 0; i < Array; i++){
        if(gold_c[i] != c[i]){
            pass = 0;
            break;
        }
    }*/

    if(c[0] != sum){
        pass =0;
    }

    if(pass==1)
        printf("test pass!\n");
    else
        printf("error...\n");
    
    return 0;

}










