#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

 __global__ void add(int *d_a, int *d_b, int *d_c){
	*d_c = *d_a + *d_b;
 }

 int main(){
	int a, b, c;
	int *d_a, *d_b, *d_c;

	hipMalloc((void**)&d_a, sizeof(int));
	hipMalloc((void**)&d_b, sizeof(int));
	hipMalloc((void**)&d_c, sizeof(int));
	a = 7;
	b = 2;
	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
	
	add<<<1, 1>>>(d_a, d_b, d_c);

	hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("%d + %d = %d\n", a, b, c);

	return 0;
 }
