#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "opencv/cv.h"
#include "opencv/highgui.h"


#define VALUE_MAX 10000000
#define WIDTH 32


struct match{
    int bestRow;
    int bestCol;
    int bestSAD;
}position;


__global__ void TM_Kernel(char *Sd_imgData, char *Pd_imgData, int *d_host_result, int R_height, int R_width, int P_height, int P_width, int S_height, int S_width, int S_widthStep, int P_widthStep);
void template_matching(char *S_imgData, char *P_imgData, int *host_result, int R_height, int R_width, int P_height, int P_width, int S_height, int S_width, int S_widthStep, int P_widthStep, int width);

//void img_info(IplImage *imgFile);

int main(int argc, char** argv){
    IplImage* sourceImg; 
    IplImage* patternImg; 
    
    int minSAD = VALUE_MAX;
    int x, y;
    int width = WIDTH;
    
    //char* ptr;

    // img data
    //char p_sourceIMG, p_patternIMG;

    //
    CvPoint pt1, pt2;
    
    // time
    // struct timespec t_start, t_end;
    // double elapsedTime;

    // shift number
    int result_height;
    int result_width;

    // difference array
    int *host_result;

    // check input files
    if(argc != 3){
        printf("Using command: %s source_image search_image\n", argv[0]);
        exit(1);
    }

    if((sourceImg = cvLoadImage( argv[1], 0)) == NULL){
        printf("%s cannot be openned\n", argv[1]);
        exit(1);
    }

    printf("height of sourceImg:%d\n", sourceImg->height);
    printf("width of sourceImg:%d\n", sourceImg->width);
    printf("size of sourceImg:%d\n", sourceImg->imageSize);

    if((patternImg = cvLoadImage(argv[2], 0)) == NULL){
        printf("%s cannot be openned\n", argv[2]);
        exit(1);
    }

    printf("height of sourceImg:%d\n", patternImg->height);
    printf("width of sourceImg:%d\n", patternImg->width);
    printf("size of sourceImg:%d\n", patternImg->imageSize);    

    // allocate memory on CPU to store SAD results
    result_height = sourceImg->height - patternImg->height + 1;
    result_width = sourceImg->width - patternImg->width + 1;
    host_result = (int *)malloc(result_height * result_width * sizeof(int));


    /*
        GPU template_matching
    */
    template_matching(sourceImg->imageData, patternImg->imageData, host_result, result_height, result_width, patternImg->height, patternImg->width, sourceImg->height, sourceImg->width, sourceImg->widthStep, patternImg->widthStep, width);

    // find minSAD
    for(y = 0; y < result_height; y++){
        for(x = 0; x < result_width; x++){
            if(minSAD > host_result[y * result_width + x]){
                minSAD = host_result[y * result_width + x];
                // give me VALUE_MAX
                position.bestRow = y;
                position.bestCol = x;
                position.bestSAD = host_result[y * result_width + x];
            }
        }
    }
    printf("minSAD is %d\n", minSAD);

    // setup the two points for the best match
    pt1.x = position.bestCol;
    pt1.y = position.bestRow;
    pt2.x = pt1.x + patternImg->width;
    pt2.y = pt1.y + patternImg->height;

    // Draw the rectangle in the source image
    cvRectangle(sourceImg, pt1, pt2, CV_RGB(255,0,0), 3, 8, 0);
    cvNamedWindow("sourceImage", 1);
    cvShowImage("sourceImage", sourceImg);
    cvNamedWindow("patternImage", 1);
    cvShowImage("patternImage", patternImg);
    cvWaitKey(0);
     

    cvDestroyWindow("sourceImage");
    cvReleaseImage(&sourceImg);
    cvDestroyWindow("patternImage");
    cvReleaseImage(&patternImg);

    return 0;

}

__global__ void TM_Kernel(char *Sd_imgData, char *Pd_imgData, int *d_host_result, int R_height, int R_width, int P_height, int P_width, int S_height, int S_width, int S_widthStep, int P_widthStep){
    // Thread row and column within matrix
    int row = (blockIdx.y * blockDim.y) + threadIdx.y;
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int i, j;
    char P_sourceIMG, P_patternIMG;

    int SAD = 0;
    if(row < R_height && col < R_width){
        for(j = 0; j < P_height; j++){
            for(i = 0; i < P_width; i++){
                P_sourceIMG = Sd_imgData[ ((row + j) * S_widthStep) + col + i];
                P_patternIMG = Pd_imgData[j * P_widthStep + i];
                SAD += abs(P_sourceIMG - P_patternIMG);
            }
        }
        d_host_result[row * R_width + col] = SAD;
    }
    
}

/*
void img_info(IplImage *imgFile){
    printf("height of sourceImg:%d\n", imgFile->height);
    printf("width of sourceImg:%d\n", imgFile->width);
    printf("size of sourceImg:%d\n", imgFile->imageSize);
}
*/

void template_matching(char *S_imgData, char *P_imgData, int *host_result, int R_height, int R_width, int P_height, int P_width, int S_height, int S_width, int S_widthStep, int P_widthStep, int width){
    size_t size_source = S_height * S_width * sizeof(char);
    size_t size_pattern = P_height * P_width * sizeof(char);
    size_t size_d_result = R_height * R_width * sizeof(int);

    char *Sd_imgData, *Pd_imgData;
    int *d_host_result;

    // Allocate
    hipMalloc((void **)&Sd_imgData, size_source);
    hipMemcpy(Sd_imgData, S_imgData, size_source, hipMemcpyHostToDevice);

    hipMalloc((void **)&Pd_imgData, size_pattern);
    hipMemcpy(Pd_imgData, P_imgData, size_pattern, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_host_result, size_d_result);

    // Setup
    int gridDim_X = (R_width + 31)/32;
    int gridDim_Y = (R_height + 31)/32;
    dim3 dimGrid(gridDim_X, gridDim_Y);
    dim3 dimBlock(width, width);

    // Get start time event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Invoke kernel
    TM_Kernel<<<dimGrid, dimBlock>>>(Sd_imgData, Pd_imgData, d_host_result, R_height, R_width, P_height, P_width, S_height, S_width, S_widthStep, P_widthStep);
    hipError_t cuda_err = hipGetLastError();
    if ( hipSuccess != cuda_err ){
        printf("before kernel call: error = %s\n", hipGetErrorString (cuda_err));
        exit(1) ;
    }

    // Get stop time event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Compute execution time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    // Read P from device memory
    hipMemcpy(host_result, d_host_result, size_d_result, hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(Sd_imgData);
    hipFree(Pd_imgData);
    hipFree(d_host_result);

}


